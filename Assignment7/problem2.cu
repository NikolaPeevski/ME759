#include "hip/hip_runtime.h"

/* Matrix Convoluion.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>
// includes, project
#include "2Dconvolution.h"

using namespace std;
////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(const Matrix M);
Matrix AllocateMatrix(int height, int width,int init);
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost);
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice);
bool CompareResults(float* A, float* B, int elements, float eps);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P);

////////////////////////////////////////////////////////////////////////////////
// Matrix convolution
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernel(Matrix M, Matrix N, Matrix P)
{

	// Your code comes here...

}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

	Matrix  M;
	Matrix  N;
	Matrix  P;


	if(argc != 2) 
	{
		printf("Usage %s Size\n",argv[0]);
		return 1;

	}

	int size = atoi(argv[1]);
	M  = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE,1);
	N  = AllocateMatrix(size, size,1);		
	P  = AllocateMatrix(size, size,0);
	float cpuTime = 0.f,gpuTime=0.f;

	// M * N on the device
//	ConvolutionOnDevice(M, N, P);

	// compute the matrix convolution on the CPU for comparison
	Matrix reference = AllocateMatrix(P.height, P.width,0);
	computeGold(reference.elements, M.elements, N.elements, N.height, N.width);

	// in this case check if the result is equivalent to the expected soluion

//	bool res = CompareResults(reference.elements, P.elements, P.width * P.height, 0.01f);;
//	if(res==0)printf("Test Failed\n"); // This shouldnt pront for correct implementation
	printf("%f\n%f\n%f\n",reference.elements[size*size-1],cpuTime,gpuTime);

	// Free matrices
	FreeMatrix(&M);
	FreeMatrix(&N);
	FreeMatrix(&P);
	return 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P)
{
	// Load M and N to the device
	Matrix Md = AllocateDeviceMatrix(M);
	CopyToDeviceMatrix(Md, M);
	Matrix Nd = AllocateDeviceMatrix(N);
	CopyToDeviceMatrix(Nd, N);

	// Allocate P on the device
	Matrix Pd = AllocateDeviceMatrix(P);
	CopyToDeviceMatrix(Pd, P);

	// Setup the execution configuration



	// Launch the device computation threads!

	// Read P from the device
	CopyFromDeviceMatrix(P, Pd); 

	// Free device matrices
	FreeDeviceMatrix(&Md);
	FreeDeviceMatrix(&Nd);
	FreeDeviceMatrix(&Pd);

}

// Allocate a device matrix of same size as M.
Matrix AllocateDeviceMatrix(const Matrix M)
{
	Matrix Mdevice = M;
	int size = M.width * M.height * sizeof(float);
	hipMalloc((void**)&Mdevice.elements, size);
	return Mdevice;
}

Matrix AllocateMatrix(int height, int width,int init) // 1 is file read/ 0 is just allocation
{
	Matrix M;
	M.width = M.pitch = width;
	M.height = height;
	int size = M.width * M.height;
	M.elements = NULL;
	FILE *fp;
	fp = fopen("problem1.inp","r");
	// don't allocate memory on option 2

	M.elements = (float*) malloc(size*sizeof(float));
	if(init)
	{
		for(unsigned int i = 0; i < M.height * M.width; i++)
		{
			fscanf(fp,"%f",&M.elements[i]);
		}
	}
	return M;
}	

// Copy a host matrix to a device matrix.
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost)
{
	int size = Mhost.width * Mhost.height * sizeof(float);
	Mdevice.height = Mhost.height;
	Mdevice.width = Mhost.width;
	Mdevice.pitch = Mhost.pitch;
	hipMemcpy(Mdevice.elements, Mhost.elements, size, 
			hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice)
{
	int size = Mdevice.width * Mdevice.height * sizeof(float);
	hipMemcpy(Mhost.elements, Mdevice.elements, size, 
			hipMemcpyDeviceToHost);
}

// Free a device matrix.
void FreeDeviceMatrix(Matrix* M)
{
	hipFree(M->elements);
	M->elements = NULL;
}

// Free a host Matrix
void FreeMatrix(Matrix* M)
{
	free(M->elements);
	M->elements = NULL;
}

//compare the data stored in two arrays on the host
bool CompareResults(float* A, float* B, int elements, float eps)
{
	for(unsigned int i = 0; i < elements; i++){
		float error = A[i]-B[i];
		if(error>eps){
			return false;
		} 
	}
	return true;
}



