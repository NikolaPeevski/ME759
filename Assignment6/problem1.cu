#include "hip/hip_runtime.h"
#define _POSIX_C_SOURCE 200809L
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <../randoms/randoms.h>


void applyStencil1D(const int sIdx, const int eIdx, const double *weights, const double *in, double *out, const int radius) {

    for(int i = sIdx; i < eIdx; i++) { // si < eIdx ???
        out[i] = 0;

        // Loop over all elements in the stencil
        for(int j = -radius; j <= radius; j++) {
            out[i] += weights[j + radius] * in[i + j];
        }
        out[i] = out[i] / (2 * radius + 1);
    }
}

void initializeWeights(double *weights, int radius) {
    int R = radius;
    double val = sqrt( 2 / (M_PI* R*R) );
    for (int i = 0; i < 2 * R + 1; i++){
        weights[i] =  val *  exp( -2*(i-R)*(i-R)/(R*R) );
    }
}


__global__ void applyStencil1D_kernel(const int sIdx, const int eIdx, const double *weights, const double *in, double *out, const int radius){
    // Define index
    int i = sIdx + blockIdx.x * blockDim.x + threadIdx.x;

    if( i < eIdx){
        out[i] = 0;
        //loop over all elements in the stencil
        for(int j = -radius; j <= radius; j++){
            out[i] += weights[j + radius] * in[i + j];
        }
        out[i] = out[i] / (2 * radius + 1);
    }
}



int main(int argc, const char * argv[]) {
    // ------------- Initializing ----------------- //
    // Get inputs
    unsigned int N = 0, seed = 0, s = 2;
    int R = 0;
    sscanf(argv[1],"%u", &N);
    sscanf(argv[2],"%u", &R);
    sscanf(argv[3],"%u", &seed);
    sscanf(argv[4],"%u", &s);
    if ( N == 0 || R == 0 || seed == 0 || s == 2){
        printf("Error in getting parameters when running the program\n\n");
        return -1;
    }
    if ( N < 2 * R + 1){
        printf("N-1 is not larger than twice the radius: increase N or decrease R.\n\n");
        return -1;
    }


    // Define sizes and weights
    int size = N * sizeof(double);
    int wsize= (2 * R + 1) * sizeof(double);
    double *weights_h;
    hipHostMalloc(&weights_h, wsize);

    initializeWeights(weights_h, R);

    // Define interval for generating random values
    int amin = -1, amax =  1;

    // Define time variables
    struct timespec start, end;

    // ------------- On device ----------------- //
    // Allocate memory on host and device
    double *in_h;
    hipHostMalloc(&in_h, size);
    double *out_h;
    hipHostMalloc(&out_h, size);
    double *in_d = NULL;
    double *out_d = NULL;
    double *weights_d = NULL;
    hipMalloc( &in_d, size);
    hipMalloc( &out_d, size);
    hipMalloc( &weights_d, size);

    // Generate random entries
    random_doubles(in_h, amin, amax, N, seed);

    // Copy context from host to device
    hipMemset(in_d, 0, size);
    hipMemset(out_d, 0, size);
    hipMemset(weights_d, 0, size);
    hipMemcpy(in_d, in_h, size, hipMemcpyHostToDevice);
    hipMemcpy(weights_d, weights_h, size, hipMemcpyHostToDevice);

    // Invoke kernel to sum arrays
    unsigned int threads = N - 2 * R;
    unsigned int blocks = (N + threads -1) / threads;

    if (s == 0){
        // No shared memory
        applyStencil1D_kernel<<<blocks, threads>>>(R, N-R, weights_d, in_d, out_d, R);
    } else {
        // Shared memory
        //vectorAdd<<<blocks,threads, threads * sizeof(int)>>>(da, dc, threads);
    }


    // Synchronize host and device
    hipDeviceSynchronize();

    // Copy c from device to host
    hipMemcpy(out_h, out_d, size, hipMemcpyDeviceToHost);


    // ------------- Compare to Stencil operation on host ----------------- //
    // Calculate on host
    double *out_href = (double*) calloc(N, sizeof(double));
    applyStencil1D(R, N-R, weights_h, in_h, out_href, R);

    // Compare
    double norm;
    for (int i = R; i < N-R; i++){
        //for (int i = 0; i < N; i++){
        norm += fabs(out_h[i] - out_href[i]);
        //printf("o:  %2.5lf\n", out_h[i]);
        //printf("oh: %2.5lf\n", out_href[i]);
    }


    // Print
    double norm2;
    for (int i = R; i < N-R; i++){
        norm2 += out_h[i] * out_h[i];
    }
    printf("%f\n", sqrt(norm2)); // Print the 2-norm

    // ------------- Free resources ----------------- //
    hipFree(in_d);
    hipFree(out_d);
    hipFree(weights_d);
    hipFree(in_h); hipFree(out_h); hipFree(weights_h);
    free(out_href);



    // ------------- TESTING ----------------- //

    printf("blablal");
    printf("Differnce: %lf\n\n", norm);

    return 0;
}