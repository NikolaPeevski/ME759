#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <randoms.h>
#include <math.h>

__global__ void sum(float* arrA, float* arrB, float* arrC, int rowSize, int colSize) {
	extern __shared__ float data[];

	data[threadIdx.x] = arrA[(blockIdx.x * rowSize) + threadIdx.x] * arrB[threadIdx.x];

	__syncthreads();	
	for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
  	{
    		if(threadIdx.x < offset) {
      			data[threadIdx.x] += data[threadIdx.x + offset];
    		}
    		__syncthreads();
  	}
 	 if(threadIdx.x == 0) {
	    arrC[blockIdx.x] = data[0];
  	}
}

int main(int argc, char *argv[]) {
    if (argv[1] == NULL) {
        printf("Wrong input");
        return 0;
    }

    int rowSize = 16;
    int ColumnSize = 32;

    unsigned int seed = 0; 
    sscanf(argv[1], "%u", &seed);

    float *hA = (float*)malloc(rowSize * ColumnSize * sizeof(float));
    float *hb = (float*)malloc(ColumnSize * sizeof(float));;
    float *hC = (float*)malloc(rowSize * sizeof(float));;

    float *dA;
    float *db;
    float *dC;

    random_floats(hA, -10, 10, rowSize * ColumnSize, seed);
    random_floats(hb, -10, 10, ColumnSize, seed);

    hipMalloc(&dA, rowSize * ColumnSize * sizeof(float));
    hipMalloc(&db, ColumnSize * sizeof(float));
    hipMalloc(&dC, rowSize * sizeof(float));

    hipMemcpy(dA, hA, rowSize * ColumnSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, ColumnSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dC, hC, rowSize * sizeof(float), hipMemcpyHostToDevice);
 
    sum<<<rowSize, ColumnSize, ColumnSize * sizeof(float)>>>(dA, db, dC, rowSize, ColumnSize);
    hipDeviceSynchronize();

    hipMemcpy(hC, dC, rowSize * sizeof(float), hipMemcpyDeviceToHost);
    double normD = 0;
    for (int i = 0; i < rowSize; ++i) {
        normD += hC[i] * hC[i];
    }

    normD = sqrt(normD);

    for (int i = 0; i < rowSize; ++i) {
        hC[i] = 0;
        for (int j = 0; j < ColumnSize; ++j) {
            hC[i] += hA[(rowSize * i) + j] * hb[j];
        }
    }

    double normH = 0;
    for (int i = 0; i < rowSize; ++i) {
        normH += hC[i] * hC[i];
    }
    normH = sqrt(normH);
    printf("%f\n", normH - normD);

    free(hA);
    free(hb);
    free(hC);
    hipFree(dA);
    hipFree(db);
    hipFree(dC);
}
