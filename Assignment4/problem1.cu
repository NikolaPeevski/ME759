
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__ void sum(int* arr, int threadSize) {
    arr[(blockIdx.x * threadSize) + threadIdx.x] = blockIdx.x + threadIdx.x;
}

int main() {

    const size_t blockSize = 2;
    const size_t threadSize = 8;
    int* dArray;
    int *hostArray = (int*)malloc(blockSize * threadSize * sizeof(int));
    hipMalloc(&dArray, blockSize * threadSize * sizeof(int));

    sum<<<2, 8>>>(dArray, threadSize);
    hipDeviceSynchronize();
    hipMemcpy(hostArray, dArray, blockSize * threadSize * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < threadSize*blockSize; i++) {
	printf("%d\n", hostArray[i]);
	}

    free(dArray);
    free(hostArray);
}
