#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <randoms.h>
#include <math.h>

#include <time.h>

__global__ void sum(double* arrA, double* arrB, double* arrC, int threadCount) {
        int i = (blockIdx.x * threadCount) + threadIdx.x;
	double sum = arrA[i] + arrB[i];
        arrC[i] = sum * sum;
}

int main(int argc, char *argv[]) {
    if (argv[1] == NULL || argv[2] == NULL || argv[3] == NULL) {
        printf("Wrong input");
        return 0;
    }
    struct timespec inclusive_start;
    struct timespec inclusive_end;

    struct timespec exclusive_start;
    struct timespec exclusive_end;

    size_t size = 0;
    sscanf(argv[1], "%zu", &size);

    int threadCount = 0;
    sscanf(argv[2], "%d", &threadCount);

    unsigned seed = 0;
    sscanf(argv[3], "%u", &seed);

    double *hA = (double*)malloc(size * sizeof(double));
    double *hB = (double*)malloc(size * sizeof(double));
    double *hC = (double*)malloc(size * sizeof(double));

    double *dA;
    double *dB;
    double *dC;

    random_doubles(hA, -1, 1, size, seed);
    random_doubles(hB, -1, 1, size, seed);

    hipMalloc(&dA, size * sizeof(double));
    hipMalloc(&dB, size * sizeof(double));
    hipMalloc(&dC, size * sizeof(double));

    //Start inclusive timing
    clock_gettime(CLOCK_MONOTONIC, &inclusive_start);
    hipMemcpy(dA, hA, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size * sizeof(double), hipMemcpyHostToDevice);

    //Start exclusive timing
    clock_gettime(CLOCK_MONOTONIC, &exclusive_start);
    sum<<<(size + threadCount - 1) / threadCount, threadCount>>>(dA, dB, dC, threadCount);
    hipDeviceSynchronize();

    //Stop exclusive timing
    clock_gettime(CLOCK_MONOTONIC, &exclusive_end);
    hipMemcpy(hC, dC, size * sizeof(double), hipMemcpyDeviceToHost);
    //Stop inclusive timing

    clock_gettime(CLOCK_MONOTONIC, &inclusive_end);
    double norm = 0;
    for (int i = 0; i < size; ++i) {
	    norm += hC[i];
    }

    norm = sqrt(norm);

    size_t inclusive_duration_usec = (inclusive_end.tv_sec - inclusive_start.tv_sec) * 1000 * 1000;

    inclusive_duration_usec += (inclusive_end.tv_nsec - inclusive_start.tv_nsec) / 1000;

    size_t exclusive_duration_usec = (exclusive_end.tv_sec - exclusive_start.tv_sec) * 1000 * 1000;

    exclusive_duration_usec += (exclusive_end.tv_nsec - exclusive_start.tv_nsec) / 1000;

    printf("%u\n", size);
    printf("%d\n", threadCount);
    printf("%zu \n", exclusive_duration_usec);
    printf("%zu \n", inclusive_duration_usec);
    printf("%f", norm);
    
    free(hA);
    free(hB);
    free(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}
