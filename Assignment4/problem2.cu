#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <randoms.h>
#include <math.h>

__global__ void sum(double* arrA, double* arrB, double* arrC, size_t size) {
    for (int i = 0; i < size; ++i) {
        double sum = arrA[i] + arrB[i];
        arrC[i] = sum * sum;
    }
}

int main(int argc, char *argv[]) {
    if (argv[1] == NULL || argv[2] == NULL || argv[3] == NULL) {
        printf("Wrong input");
        return 0;
    }
    size_t size = 0;
    sscanf(argv[1], "%zu", &size);

    int threadCount = 0;
    sscanf(argv[2], "%d", &threadCount);

    unsigned seed = 0;
    sscanf(argv[3], "%u", &seed);

    double *hA = (double*)malloc(size * sizeof(double));
    double *hB = (double*)malloc(size * sizeof(double));
    double *hC = (double*)malloc(size * sizeof(double));

    double *dA;
    double *dB;
    double *dC;

    random_doubles(hA, -1, 1, size, seed);
    random_doubles(hB, -1, 1, size, seed);

    hipMalloc(&dA, size * sizeof(double));
    hipMalloc(&dB, size * sizeof(double));
    hipMalloc(&dC, size * sizeof(double));

    //Start inclusive timing
    hipMemcpy(dA, hA, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size * sizeof(double), hipMemcpyHostToDevice);

    //Start exclusive timing
    sum<<<1, threadCount>>>(dA, dB, dC, size);
    hipDeviceSynchronize();

    //Stop exclusive timing

    hipMemcpy(hC, dC, size * sizeof(double), hipMemcpyDeviceToHost);
    //Stop inclusive timing
    double norm = 0;
    for (int i = 0; i < size; ++i) {
#        printf("%f\n", hC[i]);
	norm += hC[i];
    }

    norm = sqrt(norm);

    printf("%u\n", size);
    printf("%d\n", threadCount);
    printf("Exclusive timing.. Add it pls\n");
    printf("Inclusive timing.. Add it pls\n");
    printf("Inclusive timing.. Add it pls\n");
    printf("%f", norm);
    
    free(hA);
    free(hB);
    free(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}
