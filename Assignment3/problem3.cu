
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    print_kernel<<<1, 4>>>();
    hipDeviceSynchronize();
}
