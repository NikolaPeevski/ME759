#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <../randoms/randoms.h>
#include <math.h>

__global__ void sum(int* arrA, int* res, int rowSize, int colSize) {

	atomicAdd(&res[0],arrA[(blockIdx.x * colSize) + threadIdx.x]);
}

int main(int argc, char *argv[]) {
    if (argv[1] == NULL) {
        printf("Wrong input");
        return 0;
    }

    int rowSize = 32;
    int ColumnSize = 64;

    unsigned int seed = 0; 
    sscanf(argv[1], "%u", &seed);

    int *hA = (int*)malloc(rowSize * ColumnSize * sizeof(int));
    int *hb = (int*)malloc(sizeof(int));

    int *dA;
    int *db;

    random_ints(hA, -10, 10, rowSize * ColumnSize, seed);

    hipMalloc(&dA, rowSize * ColumnSize * sizeof(int));
    hipMalloc(&db, sizeof(int));

    hipMemcpy(dA, hA, rowSize * ColumnSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, sizeof(int), hipMemcpyHostToDevice);
 
    sum<<<rowSize, ColumnSize>>>(dA, db, rowSize, ColumnSize);
    hipDeviceSynchronize();

    hipMemcpy(hb, db, sizeof(int), hipMemcpyDeviceToHost);
    printf ("%d\n", hb[0]);

    free(hA);

    hipFree(dA);
    hipFree(db);
}
